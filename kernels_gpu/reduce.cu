#include "hip/hip_runtime.h"
/*
Copyright 2017 the anode authors

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/

#include "lib.h"
#include "common.cuh"
#include "common_reduce.cuh"
#include <hip/hip_runtime.h>

template <typename T, typename Reduce>
__global__ void anode_gpu_reduce_I1ab_Ob_packed_deterministic_kernel(
    uint32_t inner_dim,
    uint32_t keep_dim,
    const T *x,
    T *y)
{
  __shared__ T cache[512];
  for (uint32_t blk = gblock(); blk < keep_dim; blk += gblockcount()) {
    T accumulator = Reduce::InitVal();
    uint32_t rdup_inner_dim = (inner_dim + blockDim.x - 1) / blockDim.x * blockDim.x;
    uint32_t tid = threadIdx.x;
    uint32_t base_tid = 0;
    for ( ; tid < rdup_inner_dim; tid += blockDim.x, base_tid += blockDim.x) {
      uint32_t i = tid - base_tid;
      if (tid < inner_dim) {
        cache[i] = x[Index2::Pack(tid, inner_dim, blk, keep_dim)];
      } else {
        cache[i] = Reduce::InitVal();
      }
      __syncthreads();
      threadblock_reduce1024<T, Reduce>(cache);
      if (0 == i) {
        Reduce::Reduce(&accumulator, cache[0]);
      }
      __syncthreads();
    }
    y[blk] = accumulator;
  }
}

extern "C" void anode_gpu_sum_reduce_I1ab_Ob_packed_deterministic_f32(
    uint32_t inner_dim,
    uint32_t keep_dim,
    const float *x,
    float *y,
    KernelConfig cfg,
    hipStream_t stream)
{
  (void)cfg;
  anode_gpu_reduce_I1ab_Ob_packed_deterministic_kernel<float, AddReduce<float>><<<keep_dim, 512, 0, stream>>>(
      inner_dim, keep_dim, x, y);
}

template <typename T, typename AtomicReduce>
__global__ void anode_gpu_reduce_I1ab_Ob_packed_atomic_accumulate_kernel(
    uint32_t inner_dim,
    uint32_t keep_dim,
    const T *x,
    T *y)
{
  __shared__ T cache[512];
  for (uint32_t blk = gblock(); blk < keep_dim; blk += gblockcount()) {
    T accumulator = AtomicReduce::InitVal();
    uint32_t rdup_inner_dim = (inner_dim + blockDim.x - 1) / blockDim.x * blockDim.x;
    uint32_t tid = threadIdx.x;
    uint32_t base_tid = 0;
    for ( ; tid < rdup_inner_dim; tid += blockDim.x, base_tid += blockDim.x) {
      uint32_t i = tid - base_tid;
      if (tid < inner_dim) {
        cache[i] = x[Index2::Pack(tid, inner_dim, blk, keep_dim)];
      } else {
        cache[i] = AtomicReduce::InitVal();
      }
      __syncthreads();
      threadblock_reduce1024<T, AtomicReduce>(cache);
      if (0 == i) {
        AtomicReduce::AtomicReduce(&accumulator, cache[0]);
      }
      __syncthreads();
    }
    AtomicReduce::AtomicReduce(&y[blk], accumulator);
  }
}
