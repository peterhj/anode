#include "hip/hip_runtime.h"
/*
Copyright 2017 the anode authors

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/

#include "lib.h"
#include "common.cuh"
#include <hip/hip_runtime.h>
#include <hip/hip_math_constants.h>

template <typename T>
class SetConstantFlatMap {
public:
  __forceinline__ __device__ static void set_constant_flat_map_idx(uint32_t idx, T c, T *y) {
    y[idx] = c;
  }
};

template <typename T>
class MultConstantFlatMap {
public:
  __forceinline__ __device__ static void constant_flat_map_idx(uint32_t idx, T c, const T *x, T *y) {
    y[idx] = c * x[idx];
  }
};

template <typename T>
class CopyFlatMap {
public:
  __forceinline__ __device__ static void FlatMapIndex(uint32_t idx, const T *x, T *y) {
    y[idx] = x[idx];
  }
};

template <typename T>
class ModulusFlatMap {
public:
  __forceinline__ __device__ static void FlatMapIndex(uint32_t idx, const T *x, T *y);
};

template <>
class ModulusFlatMap<float> {
public:
  __forceinline__ __device__ static void FlatMapIndex(uint32_t idx, const float *x, float *y) {
    y[idx] = fabsf(x[idx]);
  }
};

template <typename T>
class SquareFlatMap {
public:
  __forceinline__ __device__ static void FlatMapIndex(uint32_t idx, const T *x, T *y) {
    T x_i = x[idx];
    y[idx] = x_i * x_i;
  }
};

template <typename T>
class PositiveClipFlatMap {
public:
  __forceinline__ __device__ static void FlatMapIndex(uint32_t idx, const T *x, T *y);
};

template <>
class PositiveClipFlatMap<float> {
public:
  __forceinline__ __device__ static void FlatMapIndex(uint32_t idx, const float *x, float *y) {
    float x_i = x[idx];
    y[idx] = x_i * static_cast<float>(x_i > 0.0f);
  }
};

template <typename T>
class UnitStepFlatMap {
public:
  __forceinline__ __device__ static void FlatMapIndex(uint32_t idx, const T *x, T *y);
};

template <>
class UnitStepFlatMap<float> {
public:
  __forceinline__ __device__ static void FlatMapIndex(uint32_t idx, const float *x, float *y) {
    float x_i = x[idx];
    y[idx] = static_cast<float>(x_i > 0.0f);
  }
};

template <typename T>
class LogPositiveClipFlatMap {
public:
  __forceinline__ __device__ static void FlatMapIndex(uint32_t idx, const T *x, T *y);
};

template <>
class LogPositiveClipFlatMap<float> {
public:
  __forceinline__ __device__ static void FlatMapIndex(uint32_t idx, const float *x, float *y) {
    float x_i = x[idx];
    if (x_i > 0.0f) {
      y[idx] = logf(x_i);
    } else {
      y[idx] = -HIP_INF_F;
    }
  }
};

template <typename T>
class PositiveReciprocalFlatMap {
public:
  __forceinline__ __device__ static void FlatMapIndex(uint32_t idx, const T *x, T *y);
};

template <>
class PositiveReciprocalFlatMap<float> {
public:
  __forceinline__ __device__ static void FlatMapIndex(uint32_t idx, const float *x, float *y) {
    float x_i = x[idx];
    if (x_i > 0.0f) {
      y[idx] = 1.0f / x_i;
    } else {
      y[idx] = 0.0f;
    }
  }
};

template <typename T>
class NormalCDFFlatMap {
public:
  __forceinline__ __device__ static void FlatMapIndex(uint32_t idx, const T *x, T *y);
};

template <>
class NormalCDFFlatMap<float> {
public:
  __forceinline__ __device__ static void FlatMapIndex(uint32_t idx, const float *x, float *y) {
    float x_i = x[idx];
    y[idx] = 0.5f * (1.0f + erff(x_i * const_rsqrt_2_f()));
  }
};

template <typename T>
class TanhFlatMap {
public:
  __forceinline__ __device__ static void FlatMapIndex(uint32_t idx, const T *x, T *y);
};

template <>
class TanhFlatMap<float> {
public:
  __forceinline__ __device__ static void FlatMapIndex(uint32_t idx, const float *x, float *y) {
    y[idx] = tanhf(x[idx]);
  }
};

template <typename T>
class Rcosh2FlatMap {
public:
  __forceinline__ __device__ static void FlatMapIndex(uint32_t idx, const T *x, T *y);
};

template <>
class Rcosh2FlatMap<float> {
public:
  __forceinline__ __device__ static void FlatMapIndex(uint32_t idx, const float *x, float *y) {
    float x_i = x[idx];
    float chx_i = coshf(x_i);
    y[idx] = 1.0f / (chx_i * chx_i);
  }
};

template <typename T, typename Map>
__global__ void anode_gpu_generic_flat_map_kernel(
    uint32_t len,
    const T *x,
    T *y)
{
  for (uint32_t idx = gtindex(); idx < len; idx += gtcount()) {
    Map::FlatMapIndex(idx, x, y);
  }
}

extern "C" void anode_gpu_copy_flat_map_f32(
    uint32_t len,
    const float *x,
    float *y,
    KernelConfig cfg,
    hipStream_t stream)
{
  anode_gpu_generic_flat_map_kernel<float, CopyFlatMap<float>><<<cfg.flat_grid_dim(len), cfg.flat_block_dim(), 0, stream>>>(
      len, x, y);
}

extern "C" void anode_gpu_modulus_flat_map_f32(
    uint32_t len,
    const float *x,
    float *y,
    KernelConfig cfg,
    hipStream_t stream)
{
  anode_gpu_generic_flat_map_kernel<float, ModulusFlatMap<float>><<<cfg.flat_grid_dim(len), cfg.flat_block_dim(), 0, stream>>>(
      len, x, y);
}

extern "C" void anode_gpu_square_flat_map_f32(
    uint32_t len,
    const float *x,
    float *y,
    KernelConfig cfg,
    hipStream_t stream)
{
  anode_gpu_generic_flat_map_kernel<float, SquareFlatMap<float>><<<cfg.flat_grid_dim(len), cfg.flat_block_dim(), 0, stream>>>(
      len, x, y);
}

extern "C" void anode_gpu_positive_clip_flat_map_f32(
    uint32_t len,
    const float *x,
    float *y,
    KernelConfig cfg,
    hipStream_t stream)
{
  anode_gpu_generic_flat_map_kernel<float, PositiveClipFlatMap<float>><<<cfg.flat_grid_dim(len), cfg.flat_block_dim(), 0, stream>>>(
      len, x, y);
}

extern "C" void anode_gpu_unit_step_flat_map_f32(
    uint32_t len,
    const float *x,
    float *y,
    KernelConfig cfg,
    hipStream_t stream)
{
  anode_gpu_generic_flat_map_kernel<float, UnitStepFlatMap<float>><<<cfg.flat_grid_dim(len), cfg.flat_block_dim(), 0, stream>>>(
      len, x, y);
}

extern "C" void anode_gpu_normal_cdf_flat_map_f32(
    uint32_t len,
    const float *x,
    float *y,
    KernelConfig cfg,
    hipStream_t stream)
{
  anode_gpu_generic_flat_map_kernel<float, NormalCDFFlatMap<float>><<<cfg.flat_grid_dim(len), cfg.flat_block_dim(), 0, stream>>>(
      len, x, y);
}

extern "C" void anode_gpu_tanh_flat_map_f32(
    uint32_t len,
    const float *x,
    float *y,
    KernelConfig cfg,
    hipStream_t stream)
{
  anode_gpu_generic_flat_map_kernel<float, TanhFlatMap<float>><<<cfg.flat_grid_dim(len), cfg.flat_block_dim(), 0, stream>>>(
      len, x, y);
}

extern "C" void anode_gpu_rcosh2_flat_map_f32(
    uint32_t len,
    const float *x,
    float *y,
    KernelConfig cfg,
    hipStream_t stream)
{
  anode_gpu_generic_flat_map_kernel<float, Rcosh2FlatMap<float>><<<cfg.flat_grid_dim(len), cfg.flat_block_dim(), 0, stream>>>(
      len, x, y);
}
