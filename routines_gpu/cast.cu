#include "hip/hip_runtime.h"
/*
Copyright 2018 Peter Jin

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/

#include "lib.h"
#include "common.cuh"
#include <cassert>
#include <hip/hip_runtime.h>
#include <hip/hip_math_constants.h>

__global__ void anode_gpu_upcast_u8_packed_u32_kernel(
    uint32_t len,
    const uint8_t *x,
    uint32_t *y)
{
  for (uint32_t idx = gtindex(); idx < len; idx += gtcount()) {
    y[idx] = ((uint32_t)(x[idx]));
  }
}

extern "C" void anode_gpu_upcast_u8_packed_u32(
    uint32_t len,
    const uint8_t *x,
    uint32_t *y,
    const KernelConfig *cfg,
    hipStream_t stream)
{
  anode_gpu_upcast_u8_packed_u32_kernel<<<cfg->flat_grid_dim(len), cfg->flat_block_dim(), 0, stream>>>(
      len, x, y);
}
