#include "hip/hip_runtime.h"
/*
Copyright 2017 the anode authors

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/

#include "lib.h"
#include "common.cuh"
#include "common_reduce.cuh"
#include <cassert>
#include <hip/hip_runtime.h>

template <typename T>
class CopyMap {
public:
  __forceinline__ __device__ static T Map(T x) {
    return x;
  }
};

template <typename T>
class SquareMap {
public:
  __forceinline__ __device__ static T Map(T x) {
    return x * x;
  }
};

template <typename T, typename Map, typename Reduce>
__global__ void anode_gpu_map_reduce_Iab_Ob_packed_deterministic_kernel(
    uint32_t reduce_dim,
    uint32_t outer_dim,
    const T *x,
    T *y)
{
  extern __shared__ T cache[];
  for (uint32_t blk = gblock(); blk < outer_dim; blk += gblockcount()) {
    T accumulator = Reduce::InitVal();
    uint32_t rdup_reduce_dim = (reduce_dim + blockDim.x - 1) / blockDim.x * blockDim.x;
    for (uint32_t i = threadIdx.x; i < rdup_reduce_dim; i += blockDim.x) {
      if (i < reduce_dim) {
        cache[threadIdx.x] = Map::Map(x[Index2::Pack(i, reduce_dim, blk)]);
      } else {
        cache[threadIdx.x] = Reduce::InitVal();
      }
      __syncthreads();
      threadblock_reduce_sync<T, Reduce>(cache);
      if (0 == threadIdx.x) {
        Reduce::Reduce(&accumulator, cache[0]);
      }
      __syncthreads();
    }
    y[blk] = accumulator;
  }
}

extern "C" void anode_gpu_sum_Iab_Ob_packed_deterministic_f32(
    uint32_t reduce_dim,
    uint32_t outer_dim,
    const float *x,
    float *y,
    const KernelConfig *cfg,
    hipStream_t stream)
{
  assert(check_power_of_2(cfg->flat_block_dim().x));
  anode_gpu_map_reduce_Iab_Ob_packed_deterministic_kernel<float, CopyMap<float>, AddReduce<float>><<<cfg->flat_block_count(outer_dim), cfg->flat_block_dim(), cfg->flat_block_len() * sizeof(float), stream>>>(
      reduce_dim, outer_dim, x, y);
}

template <typename T, typename Map, typename Reduce>
__global__ void anode_gpu_map_reduce_Iabc_Ob_packed_deterministic_kernel(
    uint32_t reduce_inner_dim,
    uint32_t mid_dim,
    uint32_t reduce_outer_dim,
    const T *x,
    T *y)
{
  extern __shared__ T cache[];
  for (uint32_t blk = gblock(); blk < mid_dim; blk += gblockcount()) {
    T accumulator = Reduce::InitVal();
    for (uint32_t j = 0; j < reduce_outer_dim; ++j) {
      uint32_t rdup_reduce_inner_dim = (reduce_inner_dim + blockDim.x - 1) / blockDim.x * blockDim.x;
      for (uint32_t i = threadIdx.x; i < rdup_reduce_inner_dim; i += blockDim.x) {
        if (i < reduce_inner_dim) {
          cache[threadIdx.x] = Map::Map(x[Index3::Pack(i, reduce_inner_dim, blk, mid_dim, j)]);
        } else {
          cache[threadIdx.x] = Reduce::InitVal();
        }
        __syncthreads();
        threadblock_reduce_sync<T, Reduce>(cache);
        if (0 == threadIdx.x) {
          Reduce::Reduce(&accumulator, cache[0]);
        }
        __syncthreads();
      }
    }
    y[blk] = accumulator;
  }
}

extern "C" void anode_gpu_sum_Iabc_Ob_packed_deterministic_f32(
    uint32_t reduce_inner_dim,
    uint32_t mid_dim,
    uint32_t reduce_outer_dim,
    const float *x,
    float *y,
    const KernelConfig *cfg,
    hipStream_t stream)
{
  assert(check_power_of_2(cfg->flat_block_dim().x));
  anode_gpu_map_reduce_Iabc_Ob_packed_deterministic_kernel<float, CopyMap<float>, AddReduce<float>><<<cfg->flat_block_count(mid_dim), cfg->flat_block_dim(), cfg->flat_block_len() * sizeof(float), stream>>>(
      reduce_inner_dim, mid_dim, reduce_outer_dim, x, y);
}

extern "C" void anode_gpu_square_map_sum_Iabc_Ob_packed_deterministic_f32(
    uint32_t reduce_inner_dim,
    uint32_t mid_dim,
    uint32_t reduce_outer_dim,
    const float *x,
    float *y,
    const KernelConfig *cfg,
    hipStream_t stream)
{
  assert(check_power_of_2(cfg->flat_block_dim().x));
  anode_gpu_map_reduce_Iabc_Ob_packed_deterministic_kernel<float, SquareMap<float>, AddReduce<float>><<<cfg->flat_block_count(mid_dim), cfg->flat_block_dim(), cfg->flat_block_len() * sizeof(float), stream>>>(
      reduce_inner_dim, mid_dim, reduce_outer_dim, x, y);
}
