#include "hip/hip_runtime.h"
/*
Copyright 2018 Peter Jin

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/

#include "lib.h"
#include "common.cuh"
#include <cassert>
#include <hip/hip_runtime.h>
#include <hip/hip_math_constants.h>

__global__ void anode_gpu_dequantize_u8_packed_kernel_f32(
    uint32_t len,
    float lo,
    float hi,
    const uint8_t *x,
    float *y)
{
  float scale = (hi - lo) / 255.0f;
  for (uint32_t idx = gtindex(); idx < len; idx += gtcount()) {
    y[idx] = ((float)(x[idx])) * scale + lo;
  }
}

extern "C" void anode_gpu_dequantize_u8_packed_f32(
    uint32_t len,
    float lo,
    float hi,
    const uint8_t *x,
    float *y,
    const KernelConfig *cfg,
    hipStream_t stream)
{
  anode_gpu_dequantize_u8_packed_kernel_f32<<<cfg->flat_grid_dim(len), cfg->flat_block_dim(), 0, stream>>>(
      len, lo, hi, x, y);
}
